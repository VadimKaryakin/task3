
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

#define CHECK(value) {                                          \
    hipError_t _m_cudaStat = value;                                        \
    if (_m_cudaStat != hipSuccess) {                                       \
        cout<< "Error:" << hipGetErrorString(_m_cudaStat) \
            << " at line " << __LINE__ << " in file " << __FILE__ << "\n"; \
        exit(1);                                                            \
    } }

__global__ void sum(float *a, float *b, float *c, int N)
{
    int id = threadIdx.x + blockIdx.x*blockDim.x;
    int threadsNum = blockDim.x*gridDim.x;
    for (int i = id; i < N; i+=threadsNum)
        c[i] = a[i]+b[i];
}

int main(void)
{
    int N = 10*1000*1000;
    float *a, *b, *c, *c_check;

    CHECK( hipMallocManaged(&a, N*4) );
    CHECK( hipMallocManaged(&b, N*4) );
    CHECK( hipMallocManaged(&c, N*4) );
    c_check = new float[N];
    for (int i = 0; i < N; i++)
    {
        a[i] = i;
        b[i] = 2*i;
    }
    for (int i = 0; i < N; i++) c_check[i] = a[i] + b[i];

    sum<<<1024, 1024>>>(a, b, c, N);

    CHECK( hipDeviceSynchronize(); );

    // check
    for (int i = 0; i < N; i++)
        if (abs(c[i] - c_check[i]) > 1e-6)
        {
            cout << "Error in element N " << i << ": c[i] = " << c[i]
                 << " c_check[i] = " << c_check[i] << "\n";
            exit(1);
        }
    CHECK( hipFree(a) );
    CHECK( hipFree(b) );
    CHECK( hipFree(c) );
    return 0;
}
